#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <cuda_runtime_api.h>


extern "C" void add_vector_gpu( float* a, float* b, float *c, int size );
__global__ void VectorAdd( float* arrayA, float* arrayB, float* output )
{
	int idx = threadIdx.x;
	output[idx] = arrayA[idx] + arrayB[idx];
}

extern "C" void add_vector_gpu( float* a, float* b, float *c, int size );
void add_vector_gpu( float* a, float* b, float *c, int size )
{
	int	data_size = size * sizeof(float);

	// part1, allocate data on device
	float	*dev_A,	*dev_B,	*dev_C;
	hipMalloc( (void**)&dev_A, data_size );
	hipMalloc( (void**)&dev_B, data_size );
	hipMalloc( (void**)&dev_C, data_size );

	// part2, copy memory to device
	hipMemcpy( dev_A, a, data_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_B, b, data_size, hipMemcpyHostToDevice );

	// part3, run kernel
	VectorAdd<<< 1, size >>>( dev_A, dev_B, dev_C );

	// part4, copy data from device
	hipMemcpy( c, dev_C, data_size, hipMemcpyDeviceToHost );

	// part5, release data
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
}
