//#include "cuPrintf.cu"

#include <hip/hip_runtime.h>
#include <stdio.h>
extern "C" void kernel_wrapper(int *a, int *b);

__global__ void kernel(int *a, int *b){
    int tx = threadIdx.x;
//  cuPrintf("tx = %d\n", tx);
    switch( tx ){
    case 0:
        *a = *a + 10;
        break;
    case 1:
        *b = *b + 3;
        break;
    default:
        break;
    }
}

void kernel_wrapper(int *a, int *b){
//  cudaPrintfInit();
    //cuPrintf("Anything...?");
    printf("Anything...?\n");
    int *d_1, *d_2;
    dim3 threads( 2, 1 );
    dim3 blocks( 1, 1 );

    hipMalloc( (void **)&d_1, sizeof(int) );
    hipMalloc( (void **)&d_2, sizeof(int) );

    hipMemcpy( d_1, a, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( d_2, b, sizeof(int), hipMemcpyHostToDevice );

    kernel<<< blocks, threads >>>( d_1, d_2 );
    hipMemcpy( a, d_1, sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( b, d_2, sizeof(int), hipMemcpyDeviceToHost );
    printf("Output: a = %d\n", a[0]);
    hipFree(d_1);
    hipFree(d_2);

//  cudaPrintfDisplay(stdout, true);
//  cudaPrintfEnd();
}
