#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//索引用到的緒構體
struct Index{
        int block, thread;
};
extern "C" void kernel_wrapper(Index *aa, Index *bb);
void OutputSpec( const hipDeviceProp_t sDevProp );

const uint3 blockIdx;
const uint3 threadIdx;
const dim3 blockDim;
//核心:把索引寫入裝置記憶體
__global__ void prob_idx(Index id[]){
        int b=blockIdx.x;       //區塊索引
        int t=threadIdx.x;      //執行緒索引
        int n=blockDim.x;       //區塊中包含的執行緒數目
        int x=b*n+t;            //執行緒在陣列中對應的位置

        //每個執行緒寫入自己的區塊和執行緒索引.
        id[x].block=b;
        id[x].thread=t;
};

//主函式
void kernel_wrapper(Index *aa,Index *bb)
{
	int  iDeviceCount = 0;
	hipGetDeviceCount( &iDeviceCount );
        printf( "Number of GPU: %d\n", iDeviceCount );

	  for( int i = 0; i < iDeviceCount; ++ i )
	  {
    		printf( "=== Device %i ===", i );
    		hipDeviceProp_t  sDeviceProp;
    		hipGetDeviceProperties( &sDeviceProp, i );
    		OutputSpec( sDeviceProp );
  	}




        Index* d=aa;
        Index* h=bb;

        //配置裝置記憶體
        hipMalloc((void**) &d, 100*sizeof(Index));

        //呼叫裝置核心
        int g=3, b=4, m=g*b;
        prob_idx <<<g,b>>>(d);

        //下載裝置記憶體內容到主機上
        hipMemcpy(h, d, 100*sizeof(Index), hipMemcpyDeviceToHost);

        //顯示內容
        for(int i=0; i<m; i++){
            printf("h[%d]={block:%d, thread:%d}\n", i,h[i].block,h[i].thread);
        }

        //釋放裝置記憶體
        hipFree(d);
 }


void OutputSpec( const hipDeviceProp_t sDevProp )
{
  printf( "Device name: %s \n", sDevProp.name );
  printf( "Device memory: %d \n", sDevProp.totalGlobalMem );
  printf( "shared Memory per-block: %d \n", sDevProp.sharedMemPerBlock );
  printf( "Register mMemory per-block: %d \n", sDevProp.regsPerBlock );
  printf( "Warp size: %d \n", sDevProp.warpSize );
  printf( "Memory pitch: %d \n", sDevProp.memPitch );
  printf( "Constant Memory: %d \n", sDevProp.totalConstMem );
  printf( "Max thread per-block: %d \n", sDevProp.maxThreadsPerBlock );
  printf( "Max Blocks per-grid: %d \n", sDevProp.maxBlocksPerGrid );
  printf( "Max thread dim: ( %d, %d, %d ) \n", sDevProp.maxThreadsDim[0], sDevProp.maxThreadsDim[1], sDevProp.maxThreadsDim[2] );
  printf( "Max grid size: ( %d, %d, %d ) \n", sDevProp.maxGridSize[0], sDevProp.maxGridSize[1], sDevProp.maxGridSize[2] );
  printf( "Ver: %d.%d \n", sDevProp.major, sDevProp.minor );
  printf( "Clock: %d \n", sDevProp.clockRate );
  printf( "textureAlignment: %d \n", sDevProp.textureAlignment );
}





