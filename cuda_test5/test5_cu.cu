//#include "../common/common.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>

extern "C" void sumArraysOnGPU1(float*d_A, float*d_B, float*d_C, float *h_A, float *h_B, size_t nBytes, float *gpuRef, float *hostRef);
clock_t t1, t2;


__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

#define CHECK(status)									\
{														\
	if (status != 0)									\
	{													\
		std::cout << "Cuda failure: " << status;		\
		abort();										\
	}													\
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}


void sumArraysOnGPU1(float*d_A, float*d_B, float*d_C, float *h_A, float *h_B, size_t nBytes, float *gpuRef, float *hostRef)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 24;//24
    printf("Vector size %d\n", nElem);

    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 1024;//512
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);
t1=clock();

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    //sumArraysOnGPU<<<32768, block>>>(d_A, d_B, d_C, nElem);
    //CHECK(cudaDeviceSynchronize());
    hipDeviceSynchronize();
t2=clock();

  printf("sumArraysOnGPU Time elapsed %f sec\n", (double)(t2-t1)/(CLOCKS_PER_SEC));

    // check kernel error
    CHECK(hipGetLastError()) ;

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

}
